/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/

#include <math.h>
#include <vector>  
#include <iostream>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>  
#include <chrono>

static inline void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

static inline int divup(int a, int b) {
    return (a + b - 1)/b;
}


static inline int roundup(int a, int b) {
    return divup(a, b) * b;
}

/*
dim3 dimBlock(256, 1);
dim3 dimGrid(1, 1);
*/

__global__ void meanKernel(int nx, int ny, float* dataGPU, float* meanGPU) {
    int j = threadIdx.x;
    for (int c = 0; c < ny; c += 256) {
        int y = c + j;
        if (y >= ny) return;
        float sumRow = 0;
        for(int x = 0; x < nx; x++){
            sumRow += dataGPU[x + y*nx];
        }
        meanGPU[y] = sumRow / nx;
    }
}

/*
dim3 dimBlock(256, 1);
dim3 dimGrid(1, 256);
*/
__global__ void differenceKernel(int nx, int ny, float* dataGPU, float* meanGPU, float* differenceGPU) {
    int i = threadIdx.x;
    int j = blockIdx.y;
    for (int c = 0; c < ny; c += 256) {
        for (int d = 0; d < nx; d += 256) {
            int x = i + d;
            int y = j + c;
            if (x >= nx && y >= ny) return;
            if (x >= nx || y >= ny) continue;
            differenceGPU[x + y*nx] = dataGPU[x + y*nx] - meanGPU[y];
        }
    }
}

/*
dim3 dimBlock(256, 1);
dim3 dimGrid(1, 1);
*/
__global__ void correlationKernel(int nx, int ny, float* correlationTransposeOriginal, float* differenceGPU) {
    int j = threadIdx.x;

    for (int c = 0; c < ny; c += 256) {
        int y = c + j;
        if (y >= ny) return;
        float sum = 0;
        for(int x = 0; x < nx; x++){
            sum += differenceGPU[x + y*nx] * differenceGPU[x + y*nx];
        }
        float sqrtSum = sqrt(sum);
        for(int x = 0; x < nx; x++){
            double cor = differenceGPU[x + y * nx]/sqrtSum;
            correlationTransposeOriginal[y + x * ny] = cor;
        }   
    }
}

/*
dim3 dimBlock(64, 1);
dim3 dimGrid(1, nny);
*/
__global__ void paddingKernel(int nx, int ny, int nny, float* correlationTransposeOriginal, float* correlationTransposeGPU) {
    int x = threadIdx.x;
    int y = blockIdx.y;

    for (int c = 0; c < nx; c += 64) {
        int j = c + x;
        if (j >= nx) return;
        float v = (y < ny) ? correlationTransposeOriginal[ny * j + y] : 0.0;
        correlationTransposeGPU[nny * j + y] = v;
    }
}

/*
dim3 dimBlock(8, 8);
dim3 dimGrid(nny/64, nny/64);
*/
__global__ void resultKernel(int nx, int ny, int nny, float *resultGPU, float* correlationTransposeGPU){
    int iThread = threadIdx.x;
    int jThread = threadIdx.y;
    int iBlock = blockIdx.x;
    int jBlock = blockIdx.y;
    if (iBlock > jBlock) {
        for (int i = 0; i < 8; i++) {
            for (int j = 0; j < 8; j++) {
                int index_i = iBlock * 64 + i * 8 + iThread;
                int index_j = jBlock * 64 + j * 8 + jThread;
                if (index_i < ny && index_j < ny) {
                    resultGPU[ny * index_i + index_j] = 0;
                } 
            }
        }
    } else {
        float v[8][8];
        for (int i = 0; i < 8; i++) {
            for (int j = 0; j < 8; j++) {
                v[i][j] = 0.0;
            }
        }
        for (int k = 0; k < nx; k++) {
            float x[8];
            float y[8];
            for (int i = 0; i < 8; i++) {
                int index_i = iBlock * 64 + i * 8 + iThread;
                x[i] = correlationTransposeGPU[nny * k + index_i];
            }
            for (int j = 0; j < 8; j++) {
                int index_j = jBlock * 64 + j * 8 + jThread;
                y[j] = correlationTransposeGPU[nny * k + index_j];
            }
            for (int i = 0; i < 8; i++) {
                for (int j = 0; j < 8; j++) {
                    v[i][j] += x[i] * y[j];
                }
            }
        }
        
        for (int i = 0; i < 8; i++) {
            for (int j = 0; j < 8; j++) {
                int index_i = iBlock * 64 + i * 8 + iThread;
                int index_j = jBlock * 64 + j * 8 + jThread;
                if (index_i < ny && index_j < ny) {
                    resultGPU[ny * index_i + index_j] = v[i][j];
                } 
            }
        }
    }
}


/* Main CPU side function */
void correlate(int ny, int nx, const float *data, float *result) {
    int nny = roundup(ny, 64);
    // Initializing the arrays
    float* dataGPU = NULL;
    float* meanGPU = NULL;
    float* differenceGPU = NULL;
    float* correlationTransposeOriginal = NULL;
    float* correlationTransposeGPU = NULL;
    float* resultGPU = NULL;
    // Allocating the data
    CHECK(hipMalloc((void**)&dataGPU, ny * nx * sizeof(float)));
    CHECK(hipMalloc((void**)&meanGPU, ny * sizeof(float)));
    CHECK(hipMalloc((void**)&differenceGPU, ny * nx * sizeof(float)));
    CHECK(hipMalloc((void**)&correlationTransposeOriginal, nx * ny * sizeof(float)));
    CHECK(hipMalloc((void**)&correlationTransposeGPU, nx * nny * sizeof(float)));
    CHECK(hipMalloc((void**)&resultGPU, ny * ny * sizeof(float)));
    // Copying the data
    CHECK(hipMemcpy(dataGPU, data, ny * nx * sizeof(float), hipMemcpyHostToDevice));
    // Run mean kernel
    {
        dim3 dimBlock(256, 1);
        dim3 dimGrid(1, 1);
        meanKernel<<<dimGrid, dimBlock>>>(nx, ny, dataGPU, meanGPU);
        CHECK(hipGetLastError());
    }
    
    // Run difference kernel
    {
        dim3 dimBlock(256, 1);
        dim3 dimGrid(1, 256);
        differenceKernel<<<dimGrid, dimBlock>>>(nx, ny, dataGPU, meanGPU, differenceGPU);
        CHECK(hipGetLastError());
    }

    // Run correlation kernel
    {
        dim3 dimBlock(256, 1);
        dim3 dimGrid(1, 1);
        correlationKernel<<<dimGrid, dimBlock>>>(nx, ny, correlationTransposeOriginal, differenceGPU);
        CHECK(hipGetLastError());
    }    

    // Run padding kernel
    {
        dim3 dimBlock(64, 1);
        dim3 dimGrid(1, nny);
        paddingKernel<<<dimGrid, dimBlock>>>(nx, ny, nny, correlationTransposeOriginal, correlationTransposeGPU);
        CHECK(hipGetLastError());
    }

    // Run calculating results kernel
    {
        dim3 dimBlock(8, 8);
        dim3 dimGrid(nny/64, nny/64);
        resultKernel<<<dimGrid, dimBlock>>>(nx, ny, nny, resultGPU, correlationTransposeGPU);
        CHECK(hipGetLastError());
    }

    hipDeviceSynchronize();
    // Copy data back the results to the CPU side 
    CHECK(hipMemcpy(result, resultGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));
    // Releasing all memory
    CHECK(hipFree(dataGPU));
    CHECK(hipFree(meanGPU));
    CHECK(hipFree(differenceGPU));
    CHECK(hipFree(correlationTransposeOriginal));
    CHECK(hipFree(correlationTransposeGPU));
    CHECK(hipFree(resultGPU));
}

// How to run GPU code in VS Code using Aalto remote computers
// First in the Remote Explorer, we click Add New, then type in this ssh:

// ssh nguyenb5@tavi -J nguyenb5@kosh.aalto.fi 

// where tavi is the Maari computer
// List of all computers
// https://www.aalto.fi/en/services/linux-computer-names-in-it-classrooms
// Replace nguyenb5 by your Aalto username

// Inpput the password when asked
// After being connected to remote computer in a new window, type in this command to download and unzip file
// wget https://ppc-exercises.cs.aalto.fi/course/aalto2022/cp/cp5/cp5.zip unzip cp5.zip
// Finally, in the terminal, type 
// ./grading test
// for grading the tests

// ./grading test
// ./grading test-plain tests/001-small-raw.txt
// ./grading test-plain tests/120-medium-simple-memcheck.txt
// ./grading test-plain benchmarks/3.txt
// ./grading test-plain benchmarks/4a.txt
