/*
This is the function you need to implement. Quick reference:
- input rows: 0 <= y < ny
- input columns: 0 <= x < nx
- element at row y and column x is stored in data[x + y*nx]
- correlation between rows i and row j has to be stored in result[i + j*ny]
- only parts with 0 <= j <= i < ny need to be filled
*/

#include <math.h>
#include <vector>
#include <iostream>
#include <cstdlib>
#include <iostream>
#include "hip/hip_runtime.h"
#include <vector>

static inline void check(hipError_t err, const char *context)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error: " << context << ": "
                  << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

static inline int divup(int a, int b)
{
    return (a + b - 1) / b;
}

/*
static inline int roundup(int a, int b) {
    return divup(a, b) * b;
}
*/

__global__ void calculateResult(int nx, int ny, float *result, float *correlation)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= ny || j >= ny)
        return;
    float sum = 0.0;
    for (int x = 0; x < nx; x++)
    {
        sum += correlation[x + i * nx] * correlation[x + j * nx];
    }
    result[i + j * ny] = sum;
}

/*
for(int i = 0; i < ny; i++){
    for(int j = 0; j <= i; j++){
        float sum = 0;
        for(int x = 0; x < nx; x++){
            sum += correlation[x + i*nx] * correlation[x + j*nx];
        }
        result[i + j * ny] = sum;
    }
}
*/

void correlate(int ny, int nx, const float *data, float *result)
{
    std::vector<float> mean(ny);
    for (int y = 0; y < ny; y++)
    {
        float sumRow = 0;
        for (int x = 0; x < nx; x++)
        {
            sumRow += data[x + y * nx];
        }
        mean[y] = sumRow / nx;
    }

    std::vector<float> difference(ny * nx);
    for (int y = 0; y < ny; y++)
    {
        for (int x = 0; x < nx; x++)
        {
            difference[x + y * nx] = data[x + y * nx] - mean[y];
        }
    }

    std::vector<float> correlation(ny * nx);
    for (int y = 0; y < ny; y++)
    {
        float sum = 0;
        for (int x = 0; x < nx; x++)
        {
            sum += difference[x + y * nx] * difference[x + y * nx];
        }
        float sqrtSum = sqrt(sum);
        for (int x = 0; x < nx; x++)
        {
            correlation[x + y * nx] = difference[x + y * nx] / sqrtSum;
        }
    }

    // Allocate memory & copy data to GPU
    /*
    float* dGPU = NULL;
    CHECK(cudaMalloc((void**)&dGPU, n * n * sizeof(float)));
    float* rGPU = NULL;
    CHECK(cudaMalloc((void**)&rGPU, n * n * sizeof(float)));
    CHECK(cudaMemcpy(dGPU, d, n * n * sizeof(float), cudaMemcpyHostToDevice));
    */
    float *correlationGPU = NULL;
    // correlation.data() return the pointers of the vectors
    CHECK(hipMalloc((void **)&correlationGPU, ny * nx * sizeof(float)));
    float *resultGPU = NULL;
    CHECK(hipMalloc((void **)&resultGPU, ny * ny * sizeof(float)));
    CHECK(hipMemcpy(correlationGPU, correlation.data(), ny * nx * sizeof(float), hipMemcpyHostToDevice));
    /*
    // Run kernel
    dim3 dimBlock(16, 16);
    dim3 dimGrid(divup(n, dimBlock.x), divup(n, dimBlock.y));
    mykernel<<<dimGrid, dimBlock>>>(rGPU, dGPU, n);
    CHECK(cudaGetLastError());
    */
    dim3 dimBlock(16, 16);
    dim3 dimGrid(divup(ny, dimBlock.x), divup(ny, dimBlock.y));
    calculateResult<<<dimGrid, dimBlock>>>(nx, ny, resultGPU, correlationGPU);
    CHECK(hipGetLastError());
    hipDeviceSynchronize();
    /*
    // Copy data back to CPU & release memory
    CHECK(cudaMemcpy(r, rGPU, n * n * sizeof(float), cudaMemcpyDeviceToHost));
    CHECK(cudaFree(dGPU));
    CHECK(cudaFree(rGPU));
    */
    // Copy data back to CPU & release memory
    CHECK(hipMemcpy(result, resultGPU, ny * ny * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(correlationGPU));
    CHECK(hipFree(resultGPU));
}

// How to run GPU code in VS Code using Aalto remote computers
// First in the Remote Explorer, we click Add New, then type in this ssh:

// ssh nguyenb5@tavi -J nguyenb5@kosh.aalto.fi 

// where tavi is the Maari computer
// List of all computers
// https://www.aalto.fi/en/services/linux-computer-names-in-it-classrooms
// Replace nguyenb5 by your Aalto username

// Inpput the password when asked
// After being connected to remote computer in a new window, type in this command to download and unzip file
// wget https://ppc-exercises.cs.aalto.fi/course/aalto2022/cp/cp4/cp4.zip unzip cp4.zip
// Finally, in the terminal, type 
// ./grading test
// for grading the tests